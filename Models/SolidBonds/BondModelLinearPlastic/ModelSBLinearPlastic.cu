#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2020, MUSEN Development Team. All rights reserved.
   This file is part of MUSEN framework http://msolids.net/musen.
   See LICENSE file for license and warranty information. */

#include "ModelSBLinearPlastic.cuh"
#include "ModelSBLinearPlastic.h"
#include <>

__constant__ double m_vConstantModelParameters[3];
__constant__ SPBC PBC;

void CModelSBLinearPlastic::SetParametersGPU(const std::vector<double>& _parameters, const SPBC& _pbc)
{
	CUDA_MEMCOPY_TO_SYMBOL(m_vConstantModelParameters, *_parameters.data(), sizeof(double) * _parameters.size());
	CUDA_MEMCOPY_TO_SYMBOL(PBC, _pbc, sizeof(SPBC));
}

void CModelSBLinearPlastic::CalculateSBGPU(double _time, double _timeStep, const SGPUParticles& _particles, SGPUSolidBonds& _bonds)
{
	CUDA_KERNEL_ARGS2_DEFAULT(CUDA_CalcSBForce_LP_kernel,
		_time,
		_timeStep,

		_particles.AnglVels,
		_particles.Coords,
		_particles.Vels,
		_particles.Forces,
		_particles.Moments,

		static_cast<unsigned>(_bonds.nElements),
		_bonds.AxialMoments,
		_bonds.CrossCuts,
		_bonds.Diameters,
		_bonds.InitialLengths,
		_bonds.LeftIDs,
		_bonds.RightIDs,
		_bonds.NormalStiffnesses,
		_bonds.NormalStrengths,
		_bonds.TangentialStiffnesses,
		_bonds.TangentialStrengths,

		_bonds.Activities,
		_bonds.EndActivities,
		_bonds.NormalPlasticStrains,
		_bonds.NormalMoments,
		_bonds.PrevBonds,
		_bonds.TangentialMoments,
		_bonds.TangentialOverlaps,
		_bonds.TotalForces
	);
}

void __global__ CUDA_CalcSBForce_LP_kernel(
	const double	_time,
	const double	_timeStep,

	const CVector3	_partAnglVels[],
	const CVector3	_partCoords[],
	const CVector3	_partVels[],
	CVector3		_partForces[],
	CVector3		_partMoments[],

	unsigned		_bondsNum,
	const double	_bondAxialMoments[],
	const double	_bondCrossCuts[],
	const double	_bondDiameters[],
	const double	_bondInitialLengths[],
	const unsigned	_bondLeftIDs[],
	const unsigned	_bondRightIDs[],
	const double	_bondNormalStiffnesses[],
	const double	_bondNormalStrengths[],
	const double	_bondTangentialStiffnesses[],
	const double	_bondTangentialStrengths[],

	uint8_t		_bondActivities[],
	double		_bondEndActivities[],
	double		_bondNormalPlasticStrains[],
	CVector3	_bondNormalMoments[],
	CVector3	_bondPrevBonds[],
	CVector3	_bondTangentialMoments[],
	CVector3	_bondTangentialOverlaps[],
	CVector3	_bondTotalForces[]
)
{
	for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < _bondsNum; i += blockDim.x * gridDim.x)
	{
		if (!_bondActivities[i]) continue;

		// relative angle velocity of contact partners
		CVector3 relAngleVel = _partAnglVels[_bondLeftIDs[i]] - _partAnglVels[_bondRightIDs[i]];

		// the bond in the global coordinate system
		CVector3 currentBond = GetSolidBond(_partCoords[_bondRightIDs[i]], _partCoords[_bondLeftIDs[i]], PBC);
		double dDistanceBetweenCenters = currentBond.Length();

		double dBondInitLength = _bondInitialLengths[i];

		// optimized
		CVector3 sumAngleVelocity = _partAnglVels[_bondLeftIDs[i]] + _partAnglVels[_bondRightIDs[i]];
		CVector3 relativeVelocity = _partVels[_bondLeftIDs[i]] - _partVels[_bondRightIDs[i]] - sumAngleVelocity*currentBond*0.5;

		CVector3 currentContact = currentBond / dDistanceBetweenCenters;
		CVector3 tempVector = _bondPrevBonds[i] * currentBond;

		CVector3 Phi = currentContact*(DotProduct(sumAngleVelocity, currentContact)*_timeStep*0.5);

		CMatrix3 M(	1 + tempVector.z*Phi.z + tempVector.y*Phi.y,	Phi.z - tempVector.z - tempVector.y*Phi.x,		-Phi.y - tempVector.z*Phi.x + tempVector.y,
					tempVector.z - Phi.z - tempVector.x*Phi.y,		tempVector.z*Phi.z + 1 + tempVector.x*Phi.x,	-tempVector.z*Phi.y + Phi.x - tempVector.x,
					-tempVector.y - tempVector.x*Phi.z + Phi.y,		-tempVector.y*Phi.z + tempVector.x - Phi.x,		tempVector.y*Phi.y + tempVector.x*Phi.x + 1);

		CVector3 normalVelocity = currentContact * DotProduct(currentContact, relativeVelocity);
		CVector3 tangentialVelocity = relativeVelocity - normalVelocity;

		// normal angle velocity
		CVector3 normalAngleVel = currentContact*DotProduct(currentContact, relAngleVel);
		CVector3 tangAngleVel = relAngleVel - normalAngleVel;

		// calculate the force
		double dStrainTotal = (dDistanceBetweenCenters-dBondInitLength) / dBondInitLength;
		double dKload = m_vConstantModelParameters[1];
		double dKUnload = m_vConstantModelParameters[2];
		CVector3 vNormalForce;

		if (_bondNormalStiffnesses[i] <= 2)
		{
			if (fabs(dKUnload*(dStrainTotal - _bondNormalPlasticStrains[i])) > fabs(dKload*dStrainTotal)) // loading stage - increase plastic strain
			{
				vNormalForce = currentContact * (-1 * _bondCrossCuts[i] * dKload*dStrainTotal*_bondNormalStiffnesses[i]);
				_bondNormalPlasticStrains[i] = dStrainTotal * (1 - dKload / dKUnload);
			}
			else
				vNormalForce = currentContact * (-1 * _bondCrossCuts[i] * dKUnload*_bondNormalStiffnesses[i]*(dStrainTotal - _bondNormalPlasticStrains[i]));
		}
		else
			vNormalForce = currentContact * (-1 * _bondCrossCuts[i] * _bondNormalStiffnesses[i] * dStrainTotal);

		_bondTangentialOverlaps[i] = M * _bondTangentialOverlaps[i] - tangentialVelocity * _timeStep;
		const CVector3 vTangentialForce = _bondTangentialOverlaps[i] * (_bondTangentialStiffnesses[i] * _bondCrossCuts[i] / dBondInitLength);
		const CVector3 vBondNormalMoment = M * _bondNormalMoments[i] - normalAngleVel * (_timeStep * 2 * _bondAxialMoments[i] * _bondTangentialStiffnesses[i] / dBondInitLength);
		const CVector3 vBondTangentialMoment = M * _bondTangentialMoments[i] - tangAngleVel * (_timeStep * _bondNormalStiffnesses[i] * _bondAxialMoments[i] / dBondInitLength);

		_bondNormalMoments[i] = vBondNormalMoment;
		_bondTangentialMoments[i] = vBondTangentialMoment;
		const CVector3 vUnsymMoment = currentBond*0.5 * vTangentialForce;
		_bondPrevBonds[i] = currentBond;
		_bondTotalForces[i] = vTangentialForce + vNormalForce;

		if (m_vConstantModelParameters[0]) 			// check the bond destruction
		{
			double dForceLength = vNormalForce.Length();
			if (dStrainTotal <= 0)	// compression
				dForceLength *= -1;
			const double dMaxStress = dForceLength / _bondCrossCuts[i] + _bondTangentialMoments[i].Length() * _bondDiameters[i] / (2 * _bondAxialMoments[i]);
			const double dMaxTorque = vTangentialForce.Length() / _bondCrossCuts[i] + _bondNormalMoments[i].Length() * _bondDiameters[i] / (4 * _bondAxialMoments[i]);
			if (( dMaxStress >= _bondNormalStrengths[i] ) || (dMaxTorque >= _bondTangentialStrengths[i]))
			{
				_bondActivities[i] = false;
				_bondEndActivities[i] = _time;
				continue; // if bond is broken do not apply forces and moments
			}
		}

		// apply forces and moments directly to particles, only if bond is not broken
		const CVector3 partForce = vNormalForce + vTangentialForce;
		const CVector3 partMoment1 = vBondNormalMoment + vBondTangentialMoment - vUnsymMoment;
		const CVector3 partMoment2 = vBondNormalMoment + vBondTangentialMoment + vUnsymMoment;
		CUDA_VECTOR3_ATOMIC_ADD(_partForces[_bondLeftIDs[i]], partForce);
		CUDA_VECTOR3_ATOMIC_ADD(_partMoments[_bondLeftIDs[i]], partMoment1);
		CUDA_VECTOR3_ATOMIC_SUB(_partForces[_bondRightIDs[i]], partForce);
		CUDA_VECTOR3_ATOMIC_SUB(_partMoments[_bondRightIDs[i]], partMoment2);
	}
}