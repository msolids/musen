#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2020, MUSEN Development Team. All rights reserved.
   This file is part of MUSEN framework http://msolids.net/musen.
   See LICENSE file for license and warranty information. */

#include "ModelSBAerogel.cuh"
#include "ModelSBAerogel.h"
#include <>

__constant__ double m_vConstantModelParameters[7];
__constant__ SPBC PBC;

void CModelSBAerogel::SetParametersGPU(const std::vector<double>& _parameters, const SPBC& _pbc)
{
	CUDA_MEMCOPY_TO_SYMBOL(m_vConstantModelParameters, *_parameters.data(), sizeof(double) * _parameters.size());
	CUDA_MEMCOPY_TO_SYMBOL(PBC, _pbc, sizeof(SPBC));
}

void CModelSBAerogel::CalculateSBGPU(double _time, double _timeStep, const SGPUParticles& _particles, SGPUSolidBonds& _bonds)
{
	CUDA_KERNEL_ARGS2_DEFAULT(CUDA_CalcSBForce_C1_kernel,
		_time,
		_timeStep,

		_particles.AnglVels,
		_particles.Coords,
		_particles.Vels,
		_particles.Forces,
		_particles.Moments,

		static_cast<unsigned>(_bonds.nElements),
		_bonds.AxialMoments,
		_bonds.CrossCuts,
		_bonds.InitialLengths,
		_bonds.LeftIDs,
		_bonds.RightIDs,
		_bonds.NormalStiffnesses,
		_bonds.TangentialStiffnesses,

		_bonds.Activities,
		_bonds.EndActivities,
		_bonds.NormalPlasticStrains,
		_bonds.TangentialPlasticStrains,
		_bonds.NormalMoments,
		_bonds.PrevBonds,
		_bonds.TangentialMoments,
		_bonds.TangentialOverlaps,
		_bonds.TotalForces
	);
}

void __global__ CUDA_CalcSBForce_C1_kernel(
	double	_time,
	double	_timeStep,

	const CVector3	_partAnglVels[],
	const CVector3	_partCoords[],
	const CVector3	_partVels[],
	CVector3		_partForces[],
	CVector3		_partMoments[],

	unsigned		_bondsNum,
	const double	_bondAxialMoments[],
	const double	_bondCrossCuts[],
	const double	_bondInitialLengths[],
	const unsigned	_bondLeftIDs[],
	const unsigned	_bondRightIDs[],
	const double	_bondNormalStiffnesses[],
	const double	_bondTangentialStiffnesses[],

	unsigned	_bondActivities[],
	double		_bondEndActivities[],
	double		_bondNormalPlasticStrains[],
	CVector3	_bondTangentialPlasticStrains[],
	CVector3	_bondNormalMoments[],
	CVector3	_bondPrevBonds[],
	CVector3	_bondTangentialMoments[],
	CVector3	_bondTangentialOverlaps[],
	CVector3	_bondTotalForces[]
)
{
	for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < _bondsNum; i += blockDim.x * gridDim.x)
	{
		if (!_bondActivities[i]) continue;

		// relative angle velocity of contact partners
		CVector3 relAngleVel = _partAnglVels[_bondLeftIDs[i]] - _partAnglVels[_bondRightIDs[i]];

		// the bond in the global coordinate system
		CVector3 currentBond = GetSolidBond(_partCoords[_bondRightIDs[i]], _partCoords[_bondLeftIDs[i]], PBC);
		double dDistanceBetweenCenters = currentBond.Length();

		double dBondInitLength = _bondInitialLengths[i];

		// optimized
		CVector3 sumAngleVelocity = _partAnglVels[_bondLeftIDs[i]] + _partAnglVels[_bondRightIDs[i]];
		CVector3 relativeVelocity = _partVels[_bondLeftIDs[i]] - _partVels[_bondRightIDs[i]] - sumAngleVelocity * currentBond*0.5;

		CVector3 currentContact = currentBond / dDistanceBetweenCenters;
		CVector3 tempVector = _bondPrevBonds[i] * currentBond;

		CVector3 Phi = currentContact * (DotProduct(sumAngleVelocity, currentContact)*_timeStep*0.5);

		CMatrix3 M(1 + tempVector.z*Phi.z + tempVector.y*Phi.y, Phi.z - tempVector.z - tempVector.y*Phi.x, -Phi.y - tempVector.z*Phi.x + tempVector.y,
			tempVector.z - Phi.z - tempVector.x*Phi.y, tempVector.z*Phi.z + 1 + tempVector.x*Phi.x, -tempVector.z*Phi.y + Phi.x - tempVector.x,
			-tempVector.y - tempVector.x*Phi.z + Phi.y, -tempVector.y*Phi.z + tempVector.x - Phi.x, tempVector.y*Phi.y + tempVector.x*Phi.x + 1);

		CVector3 normalVelocity = currentContact * DotProduct(currentContact, relativeVelocity);
		CVector3 tangentialVelocity = relativeVelocity - normalVelocity;

		// normal angle velocity
		CVector3 normalAngleVel = currentContact * DotProduct(currentContact, relAngleVel);
		CVector3 tangAngleVel = relAngleVel - normalAngleVel;

		// calculate the force
		double &dBroken = _bondTangentialPlasticStrains[i].x;
		double dPureStrain = (dDistanceBetweenCenters - dBondInitLength) / dBondInitLength;
		double dPlasticStrainCompr = -fabs(m_vConstantModelParameters[0]);
		double dPlasticStrainTens  = fabs(m_vConstantModelParameters[1]);
		double dSoftnessRatioCompr = m_vConstantModelParameters[2];
		double dSoftnessRatioTens  = m_vConstantModelParameters[3];
		double dBreakageStrainTens = fabs(m_vConstantModelParameters[4]);
		double dHardnessRatioCompr = m_vConstantModelParameters[5];
		double dHardnessRatioTens  = m_vConstantModelParameters[6];
		double dKA = _bondCrossCuts[i] * _bondNormalStiffnesses[i];

		double dNormalActingForce = 0.;

		if (fabs(dBroken) < 0.5)
		{
			if ((dPureStrain < 0) && (dPureStrain < dPlasticStrainCompr))
			{
				double dDeltaPlasticStrain = (dPureStrain - dPlasticStrainCompr);
				dNormalActingForce = dPlasticStrainCompr * dKA + dDeltaPlasticStrain * dSoftnessRatioCompr*dKA;
				_bondNormalPlasticStrains[i] = dDeltaPlasticStrain * (1 - dSoftnessRatioCompr);
				if (dNormalActingForce > 0)
				{
					dBroken = -1.;
					dNormalActingForce = 0.;
				}
			}
			else if ((dPureStrain > 0) && (dPureStrain > dPlasticStrainTens))
			{
				double dDeltaPlasticStrain = (dPureStrain - dPlasticStrainTens);
				dNormalActingForce = dPlasticStrainTens * dKA + dDeltaPlasticStrain * dSoftnessRatioTens*dKA;
				_bondNormalPlasticStrains[i] = dDeltaPlasticStrain * (1 - dSoftnessRatioTens);
				if (dNormalActingForce < 0)
				{
					dBroken = 1.;
					dNormalActingForce = 0.;
				}
			}
			else
				dNormalActingForce = (dPureStrain - _bondNormalPlasticStrains[i]) * dKA;
			if (dNormalActingForce  > (dBreakageStrainTens*dKA)) // breakage by tension
			{
				_bondActivities[i] = false;
				_bondEndActivities[i] = _time;
			}
		}
		else if (dBroken < -0.5)
		{
			if (dPureStrain < _bondNormalPlasticStrains[i])
			{
				_bondNormalPlasticStrains[i] = dPureStrain;
			}
			dNormalActingForce = dKA * dHardnessRatioCompr * (dPureStrain - _bondNormalPlasticStrains[i]);
		}
		else if (dBroken > 0.5)
		{
			if (dPureStrain > _bondNormalPlasticStrains[i])
			{
				_bondNormalPlasticStrains[i] = dPureStrain;
			}
			dNormalActingForce = dKA * dHardnessRatioTens * (dPureStrain - _bondNormalPlasticStrains[i]);
		}

		CVector3 vNormalForce = -1 * currentContact * dNormalActingForce;

		_bondTangentialOverlaps[i] = M * _bondTangentialOverlaps[i] - tangentialVelocity * _timeStep;
		const CVector3 vTangentialForce = _bondTangentialOverlaps[i] * (_bondTangentialStiffnesses[i] * _bondCrossCuts[i] / dBondInitLength);
		const CVector3 vBondNormalMoment = M * _bondNormalMoments[i] - normalAngleVel * (_timeStep * 2 * _bondAxialMoments[i] * _bondTangentialStiffnesses[i] / dBondInitLength);
		const CVector3 vBondTangentialMoment = M * _bondTangentialMoments[i] - tangAngleVel * (_timeStep * _bondNormalStiffnesses[i] * _bondAxialMoments[i] / dBondInitLength);

		const CVector3 vUnsymMoment = currentBond * 0.5 * vTangentialForce;

		_bondPrevBonds[i] = currentBond;
		_bondTotalForces[i] = vTangentialForce + vNormalForce;
		_bondNormalMoments[i] = vBondNormalMoment;
		_bondTangentialMoments[i] = vBondTangentialMoment;

		// apply forces and moments directly to particles
		const CVector3 partForce = vNormalForce + vTangentialForce;
		const CVector3 partMoment1 = vBondNormalMoment + vBondTangentialMoment - vUnsymMoment;
		const CVector3 partMoment2 = vBondNormalMoment + vBondTangentialMoment + vUnsymMoment;
		CUDA_VECTOR3_ATOMIC_ADD(_partForces[_bondLeftIDs[i]], partForce);
		CUDA_VECTOR3_ATOMIC_ADD(_partMoments[_bondLeftIDs[i]], partMoment1);
		CUDA_VECTOR3_ATOMIC_SUB(_partForces[_bondRightIDs[i]], partForce);
		CUDA_VECTOR3_ATOMIC_SUB(_partMoments[_bondRightIDs[i]], partMoment2);
	}
}